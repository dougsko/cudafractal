#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <gd.h>

#define DIM 1000

int
makePNG(char *imgData)
{
    gdImagePtr image;
    FILE *out;
    int color;

    image = gdImageCreate(DIM, DIM);
    color = gdImageColorAllocate(image, 0, 0, 0); // R, G, B
    gdImageSetPixel(image, 0, 0, color);
    out = fopen("foo.png", "wb");
    gdImagePng(image, out);
    fclose(out);
    gdImageDestroy(image);

    return 0;
}

struct 
hipComplex
{
    float r;
    float i;
    __device__ hipComplex(float a, float b) : r(a), i(b) {}
    
    __device__ float 
    magnatude2(void)
    {
        return r * r + i * i;
    }

    __device__ hipComplex 
    operator*(const hipComplex& a)
    {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }

    __device__ hipComplex 
    operator+(const hipComplex& a)
    {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int 
julia(int x, int y)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for(i = 0; i < 200; i++)
    {
        a = a * a + c;
        if(a.magnatude2() > 1000)
            return 0;
    }
    return 1;
}

__global__ void 
kernel(unsigned char *ptr)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    int juliaValue = julia(x, y);
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

int
main(void)
{
    unsigned char *dev_bitmap;
    unsigned char bitmap[4000];

    int i;
    for( i = 0; i <= DIM*4; i++)
    {
        bitmap[i] = 0;
    }

    hipMalloc((void **) &dev_bitmap, DIM * DIM);

    hipMemcpy(dev_bitmap, bitmap, DIM * 4 * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(DIM, DIM);
    kernel<<<grid, 1>>>(dev_bitmap);

    hipMemcpy(bitmap, dev_bitmap, DIM * 4 * sizeof(int), hipMemcpyDeviceToHost);
    for(i = 0; i <= 4000; i++)
    {
        printf("%d\n", bitmap[i]);
    }

    hipFree(dev_bitmap);

    return 0;
}

