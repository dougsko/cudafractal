#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cpu_bitmap.h"
#include "book.h"
#include "png_helper.h"

#define DIM 1080
#define DIM_X 1920
#define DIM_Y 1080


struct 
hipComplex
{
    float r;
    float i;
    __device__ hipComplex(float a, float b) : r(a), i(b) {}
    
    __device__ float 
    magnatude2(void)
    {
        return r * r + i * i;
    }

    __device__ hipComplex 
    operator*(const hipComplex& a)
    {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }

    __device__ hipComplex 
    operator+(const hipComplex& a)
    {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int 
julia(int x, int y)
{
    const float scale = 0.8;
    float jx = scale * (float)(DIM_X/2 - x)/(DIM_X/2);
    float jy = scale * (float)(DIM_Y/2 - y)/(DIM_Y/2);
    float mag;

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for(i = 0; i < 200; i++)
    {
        a = a * a + c;
        mag = a.magnatude2();
        if(mag > 1000)
            return mag;
    }
    return 1;
}

__global__ void 
kernel(unsigned char *ptr)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    int juliaValue = julia(x, y);
    if(juliaValue == 1)
        ptr[offset*4 + 0] = 255 * juliaValue;
    else
        ptr[offset*4 + 0] = 0;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

int
main(void)
{
    unsigned char *dev_bitmap;
    CPUBitmap bitmap(DIM_X, DIM_Y);


    hipMalloc((void **) &dev_bitmap, bitmap.image_size());

    dim3 grid(DIM_X, DIM_Y);
    kernel<<<grid, 1>>>(dev_bitmap);

    hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);

    bitmap.display_and_exit();

    hipFree(dev_bitmap);

    return 0;
}

